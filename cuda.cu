#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include "common.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) C[tid] = A[tid] + B[tid];
}

int main(int argc, char **argv) {
    int nElem = 1 << 28;
    if (argc > 1) nElem = 1 << atoi(argv[1]);
    size_t nBytes = nElem * sizeof(float);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    printf("Vector size: %d\n\n", nElem);

    // malloc host and unified memory
    float *A, *B, *hostRef, *gpuRef;
    CHECK(hipMallocManaged((void **)&A, nBytes));
    CHECK(hipMallocManaged((void **)&B, nBytes));
    CHECK(hipMallocManaged((void **)&hostRef, nBytes));
    CHECK(hipMallocManaged((void **)&gpuRef, nBytes));

    initialData(A, nElem);
    initialData(B, nElem);

    sumArraysOnHost(A, B, hostRef, nElem);

    dim3 block(1024);
    dim3 grid((nElem + block.x - 1) / block.x);

    // warmup
    sumArraysOnGPU<<<grid, block>>>(A, B, gpuRef, nElem);
    CHECK(hipDeviceSynchronize());

    double dtime = - omp_get_wtime();
    sumArraysOnGPU<<<grid, block>>>(A, B, gpuRef, nElem);
    CHECK(hipDeviceSynchronize());
    dtime += omp_get_wtime();
    printf("\"sumArraysOnGPU\" with <<<grid %d, block %d>>>\n", grid.x, block.x);
    printf("Elapsed time: %.3f sec, %lf GFLOPS\n\n", dtime, COST * nElem / dtime / 1.0e+9);
    checkResult(hostRef, gpuRef, nElem);

    CHECK(hipFree(A));
    CHECK(hipFree(B));
    CHECK(hipFree(hostRef));
    CHECK(hipFree(gpuRef));

    CHECK(hipDeviceReset());

    return 0;
}