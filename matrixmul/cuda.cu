#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include "matrixmul.h"

__global__ void mulMatrixOnGPU(float *A, float *B, float *C, const int N) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * N + ix;

    if (ix < N && iy < N) {
        float temp = 0.0;
        for (int i = 0; i < N; i++) {
            temp += A[iy * N + i] * B[i * N + ix];
        }
        C[idx] = temp;
    }
}

int main(int argc, char **argv) {
    int nElem = 1 << 10;
    if (argc > 1) nElem = 1 << atoi(argv[1]);
    int nxy = nElem * nElem;
    size_t nBytes = nxy * sizeof(float);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    printf("Matrix size: %d * %d\n\n", nElem, nElem);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    call_cblas_sgemm(h_A, h_B, hostRef, nElem);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float **)&d_A, nBytes));
    CHECK(hipMalloc((float **)&d_B, nBytes));
    CHECK(hipMalloc((float **)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    dim3 block(32, 32);
    if (argc > 3) {
        block.x = atoi(argv[2]);
        block.y = atoi(argv[3]);
    }
    dim3 grid((nElem + block.x - 1) / block.x, (nElem + block.y - 1) / block.y);

    // warmup
    mulMatrixOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipDeviceSynchronize());

    double dtime = - omp_get_wtime();
    mulMatrixOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    dtime += omp_get_wtime();
    printf("\"mulMatrixOnGPU\" with <<<grid (%d, %d), block (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);
    printf("Elapsed time: %.3f sec, %.4f TFLOPS\n\n", dtime, calcMmulTFLOPS(nElem, dtime));

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nxy);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    CHECK(hipDeviceReset());

    return 0;
}