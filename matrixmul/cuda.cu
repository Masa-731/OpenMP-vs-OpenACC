#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include "../common.h"

__global__ void mulMatrixOnGPU(float *A, float *B, float *C, const int N) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * N + ix;

    if (ix < N && iy < N) {
        float temp = 0.0;
        for (int i = 0; i < N; i++) {
            temp += A[iy * N + i] * B[i * N + ix];
        }
        C[idx] = temp;
    }
}

int main(int argc, char **argv) {
    int nElem = 1 << 10;
    if (argc > 1) nElem = 1 << atoi(argv[1]);
    size_t nBytes = nElem * nElem * sizeof(float);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    printf("Vector size: %d\n\n", nElem);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    if (nElem < (1 << 13)) mulMatrixOnHost(h_A, h_B, hostRef, nElem);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float **)&d_A, nBytes));
    CHECK(hipMalloc((float **)&d_B, nBytes));
    CHECK(hipMalloc((float **)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    dim3 block(32);
    dim3 grid((nElem + block.x - 1) / block.x);

    // warmup
    mulMatrixOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipDeviceSynchronize());

    double dtime = - omp_get_wtime();
    mulMatrixOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    dtime += omp_get_wtime();
    printf("\"mulMatrixOnGPU\" with <<<grid (%d, %d), block (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);
    printf("Elapsed time: %.3f msec\n\n", dtime / 1.0e-3);

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if (nElem < (1 << 13)) checkResult(hostRef, gpuRef, nElem * nElem);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    CHECK(hipDeviceReset());

    return 0;
}